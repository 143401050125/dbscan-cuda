
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <algorithm>
#include <map>
#include <set>
#include <vector>

using namespace std;

// #define DATASET_COUNT 100000
#define DATASET_COUNT 1864620
#define DIMENSION 2
#define MAX_SEEDS 1024
#define REFILL_MAX_SEEDS 2048
#define EXTRA_COLLISION_SIZE 256
#define THREAD_BLOCKS 16
#define THREAD_COUNT 1024
#define UNPROCESSED -1
#define NOISE -2
#define MINPTS 4
#define EPS 1.5

__device__ __constant__ int minPts = MINPTS;
__device__ __constant__ float eps = EPS;

#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}

int ImportDataset(char const *fname, double *dataset);
bool MonitorSeedPoints(vector<int> &unprocessedPoints, int *runningCluster,
                       int *clusterCount, int *noiseCount, int *d_cluster,
                       int *d_seedList, int *d_seedLength,
                       int *d_refillSeedList, int *d_refillSeedLength,
                       int *d_collisionMatrix, int *d_extraCollision,
                       int *d_extraCollisionLength);
__global__ void DBSCAN(double *dataset, int *cluster, int *seedList,
                       int *seedLength, int *refillSeedList,
                       int *refillSeedLength, int *collisionMatrix,
                       int *extraCollision, int *extraCollisionLength);
__device__ void MarkAsCandidate(int neighborID, int chainID, int *cluster,
                                int *seedList, int *seedLength,
                                int *refillSeedList, int *refillSeedLength,
                                int *collisionMatrix, int *extraCollision,
                                int *extraCollisionLength);
int main(int argc, char **argv) {
  /**
   **************************************************************************
   * Get the dataset file from argument and import data
   **************************************************************************
   */

  char inputFname[500];
  if (argc != 2) {
    fprintf(stderr, "Please provide the dataset file path in the arguments\n");
    exit(0);
  }

  strcpy(inputFname, argv[1]);

  printf("Using dataset file %s\n", inputFname);

  double *importedDataset =
      (double *)malloc(sizeof(double) * DATASET_COUNT * DIMENSION);

  int ret = ImportDataset(inputFname, importedDataset);

  for (int i = 0; i < 2; i++) {
    printf("Sample Data %f\n", importedDataset[i]);
  }

  if (ret == 1) {
    printf("\nError importing the dataset");
    return 0;
  }

  vector<int> unprocessedPoints;
  for (int x = 0; x < DATASET_COUNT; x++) {
    unprocessedPoints.push_back(x);
  }
  printf("Imported %lu data in dataset\n", unprocessedPoints.size());

  gpuErrchk(hipDeviceReset());
  gpuErrchk(hipFree(0));

  /**
   **************************************************************************
   * Memory allocation
   **************************************************************************
   */
  double *d_dataset;
  int *d_cluster;
  int *d_seedList;
  int *d_seedLength;
  int *d_refillSeedList;
  int *d_refillSeedLength;
  int *d_collisionMatrix;
  int *d_extraCollision;
  int *d_extraCollisionLength;

  gpuErrchk(hipMalloc((void **)&d_dataset,
                       sizeof(double) * DATASET_COUNT * DIMENSION));
  gpuErrchk(hipMalloc((void **)&d_cluster, sizeof(int) * DATASET_COUNT));
  gpuErrchk(hipMalloc((void **)&d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS));
  gpuErrchk(hipMalloc((void **)&d_seedLength, sizeof(int) * THREAD_BLOCKS));
  gpuErrchk(hipMalloc((void **)&d_refillSeedList,
                       sizeof(int) * THREAD_BLOCKS * REFILL_MAX_SEEDS));
  gpuErrchk(
      hipMalloc((void **)&d_refillSeedLength, sizeof(int) * THREAD_BLOCKS));
  gpuErrchk(hipMalloc((void **)&d_collisionMatrix,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS));
  gpuErrchk(hipMalloc((void **)&d_extraCollision,
                       sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE));
  gpuErrchk(hipMalloc((void **)&d_extraCollisionLength,
                       sizeof(int) * THREAD_BLOCKS));

  /**
   **************************************************************************
   * Assignment with default values
   **************************************************************************
   */
  gpuErrchk(hipMemcpy(d_dataset, importedDataset,
                       sizeof(double) * DATASET_COUNT * DIMENSION,
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMemset(d_cluster, UNPROCESSED, sizeof(int) * DATASET_COUNT));
  gpuErrchk(
      hipMemset(d_seedList, -1, sizeof(int) * THREAD_BLOCKS * MAX_SEEDS));
  gpuErrchk(hipMemset(d_seedLength, 0, sizeof(int) * THREAD_BLOCKS));
  gpuErrchk(hipMemset(d_refillSeedList, -1,
                       sizeof(int) * THREAD_BLOCKS * REFILL_MAX_SEEDS));
  gpuErrchk(hipMemset(d_refillSeedLength, 0, sizeof(int) * THREAD_BLOCKS));
  gpuErrchk(hipMemset(d_collisionMatrix, -1,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS));
  gpuErrchk(hipMemset(d_extraCollision, -1,
                       sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE));
  gpuErrchk(hipMemset(d_extraCollisionLength, 0, sizeof(int) * THREAD_BLOCKS));

  /**
   **************************************************************************
   * Start the DBSCAN algorithm
   **************************************************************************
   */

  int runningCluster = 0;
  int clusterCount = 0;
  int noiseCount = 0;
  bool exit = false;
  while (!exit) {
    int completed = MonitorSeedPoints(
        unprocessedPoints, &runningCluster, &clusterCount, &noiseCount,
        d_cluster, d_seedList, d_seedLength, d_refillSeedList,
        d_refillSeedLength, d_collisionMatrix, d_extraCollision,
        d_extraCollisionLength);

    if (completed) {
      exit = true;
    }

    printf("Running cluster %d, unprocessed points: %lu\n", runningCluster,
           unprocessedPoints.size());

    if (exit) break;

    gpuErrchk(hipDeviceSynchronize());

    DBSCAN<<<dim3(THREAD_BLOCKS, 1), dim3(THREAD_COUNT, 1)>>>(
        d_dataset, d_cluster, d_seedList, d_seedLength, d_refillSeedList,
        d_refillSeedLength, d_collisionMatrix, d_extraCollision,
        d_extraCollisionLength);

    gpuErrchk(hipDeviceSynchronize());
  }

  /**
   **************************************************************************
   * End DBSCAN and show the results
   **************************************************************************
   */

  printf("Final cluster after merging: %d\n", clusterCount);
  printf("Number of noises: %d\n", noiseCount);

  /**
   **************************************************************************
   * Free memory allocations
   **************************************************************************
   */
  hipFree(d_dataset);
  hipFree(d_cluster);
  hipFree(d_seedList);
  hipFree(d_seedLength);
  hipFree(d_refillSeedList);
  hipFree(d_refillSeedLength);
  hipFree(d_collisionMatrix);
  hipFree(d_extraCollision);
  hipFree(d_extraCollisionLength);
}

bool MonitorSeedPoints(vector<int> &unprocessedPoints, int *runningCluster,
                       int *clusterCount, int *noiseCount, int *d_cluster,
                       int *d_seedList, int *d_seedLength,
                       int *d_refillSeedList, int *d_refillSeedLength,
                       int *d_collisionMatrix, int *d_extraCollision,
                       int *d_extraCollisionLength) {
  /**
   **************************************************************************
   * Define and copy GPU variables to CPU variables
   **************************************************************************
   */
  int *localSeedLength;
  localSeedLength = (int *)malloc(sizeof(int) * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localSeedLength, d_seedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyDeviceToHost));

  int *localRefillSeedLength;
  localRefillSeedLength = (int *)malloc(sizeof(int) * THREAD_BLOCKS);
  gpuErrchk(hipMemcpy(localRefillSeedLength, d_refillSeedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyDeviceToHost));

  int *localSeedList;
  localSeedList = (int *)malloc(sizeof(int) * THREAD_BLOCKS * MAX_SEEDS);
  gpuErrchk(hipMemcpy(localSeedList, d_seedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyDeviceToHost));

  int *localRefillSeedList;
  localRefillSeedList =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * REFILL_MAX_SEEDS);
  gpuErrchk(hipMemcpy(localRefillSeedList, d_refillSeedList,
                       sizeof(int) * THREAD_BLOCKS * REFILL_MAX_SEEDS,
                       hipMemcpyDeviceToHost));

  /**
   **************************************************************************
   * Check if the seedlist is not empty, If so continue with DBSCAN process
   * if seedlist is empty, check refill seed list
   * if there are points in refill list, transfer to seedlist
   **************************************************************************
   */

  int completeSeedListFirst = false;
  int refilled = false;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    if (localSeedLength[i] > 0) {
      completeSeedListFirst = true;
      break;
    } else {
      if (localRefillSeedLength[i] > 0) {
        // printf("%d Refill, seedLength %d, refill Length %d \n", i,
        // localSeedLength[i], localRefillSeedLength[i]);

        while (localSeedLength[i] < MAX_SEEDS && localRefillSeedLength[i] > 0) {
          localRefillSeedLength[i] = localRefillSeedLength[i] - 1;

          localSeedList[i * MAX_SEEDS + localSeedLength[i]] =
              localRefillSeedList[i * REFILL_MAX_SEEDS +
                                  localRefillSeedLength[i]];

          localSeedLength[i] = localSeedLength[i] + 1;
        }
        // printf("%d Refill, seedLength %d, refill Length %d \n", i,
        // localSeedLength[i], localRefillSeedLength[i]);
        // printf("==========================\n");

        refilled = true;
        break;
      }
    }
  }

  /**
   **************************************************************************
   * If seedlist still have points, go to DBSCAN process
   **************************************************************************
   */

  if (completeSeedListFirst) {
    free(localSeedList);
    free(localSeedLength);
    free(localRefillSeedList);
    free(localRefillSeedLength);
    return false;
  }

  /**
   **************************************************************************
   * If refill has been done, reload the seedlist and seedlist to GPU
   * and return to DBSCAN process
   **************************************************************************
   */
  if (refilled) {
    gpuErrchk(hipMemcpy(d_seedLength, localSeedLength,
                         sizeof(int) * THREAD_BLOCKS, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_seedList, localSeedList,
                         sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                         hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_refillSeedLength, localRefillSeedLength,
                         sizeof(int) * THREAD_BLOCKS, hipMemcpyHostToDevice));
    free(localSeedList);
    free(localSeedLength);
    free(localRefillSeedList);
    free(localRefillSeedLength);
    return false;
  }

  /**
   **************************************************************************
   * Define and copy GPU variables to CPU variables
   **************************************************************************
   */

  int *localCluster;
  localCluster = (int *)malloc(sizeof(int) * DATASET_COUNT);

  gpuErrchk(hipMemcpy(localCluster, d_cluster, sizeof(int) * DATASET_COUNT,
                       hipMemcpyDeviceToHost));

  int *localCollisionMatrix;
  localCollisionMatrix =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS);

  gpuErrchk(hipMemcpy(localCollisionMatrix, d_collisionMatrix,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS,
                       hipMemcpyDeviceToHost));

  int *localExtraCollision;
  localExtraCollision =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE);

  gpuErrchk(hipMemcpy(localExtraCollision, d_extraCollision,
                       sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE,
                       hipMemcpyDeviceToHost));
  int *localExtraCollisionLength;
  localExtraCollisionLength =
      (int *)malloc(sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE);

  gpuErrchk(hipMemcpy(localExtraCollisionLength, d_extraCollisionLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyDeviceToHost));

  /**
   **************************************************************************
   * If seedlist is empty and refill is also empty
   * Then check the collision between chains
   * and finalize the clusters
   **************************************************************************
   */

  map<int, int> clusterMap;
  set<int> blockSet;
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    clusterMap[i] = i;
    blockSet.insert(i);
  }

  set<int>::iterator it;
  while (blockSet.empty() == 0) {
    it = blockSet.begin();
    int curBlock = *it;
    set<int> expansionQueue;
    set<int> finalQueue;
    expansionQueue.insert(curBlock);
    finalQueue.insert(curBlock);
    while (expansionQueue.empty() == 0) {
      it = expansionQueue.begin();
      int expandBlock = *it;
      expansionQueue.erase(it);
      blockSet.erase(expandBlock);
      for (int x = 0; x < THREAD_BLOCKS; x++) {
        if (x == expandBlock) continue;
        if ((localCollisionMatrix[expandBlock * THREAD_BLOCKS + x] == 1 ||
             localCollisionMatrix[x * THREAD_BLOCKS + expandBlock] == 1) &&
            blockSet.find(x) != blockSet.end()) {
          expansionQueue.insert(x);
          finalQueue.insert(x);
        }
      }
    }

    for (it = finalQueue.begin(); it != finalQueue.end(); ++it) {
      clusterMap[*it] = curBlock;
    }
  }

  vector<vector<int>> clustersList(THREAD_BLOCKS, vector<int>());
  for (int i = 0; i < DATASET_COUNT; i++) {
    if (localCluster[i] >= 0 && localCluster[i] < THREAD_BLOCKS) {
      clustersList[clusterMap[localCluster[i]]].push_back(i);
    }
  }

  // Check extra collision
  for (int i = 0; i < THREAD_BLOCKS; i++) {
    if (localExtraCollisionLength[i] == 0) continue;
    if (localExtraCollisionLength[i] > 1) {
      for (int j = 1; j < localExtraCollisionLength[i]; j++) {
        for (int k = 0; k < DATASET_COUNT; k++) {
          if (localCluster[k] ==
              localExtraCollision[i * EXTRA_COLLISION_SIZE + j]) {
            localCluster[k] = localExtraCollision[i * EXTRA_COLLISION_SIZE];
          }
        }
      }
    }
    for (int x = 0; x < clustersList[clusterMap[i]].size(); x++) {
      localCluster[clustersList[clusterMap[i]][x]] =
          localExtraCollision[i * EXTRA_COLLISION_SIZE];
    }
    clustersList[clusterMap[i]].clear();
  }

  for (int i = 0; i < clustersList.size(); i++) {
    if (clustersList[i].size() == 0) continue;
    for (int x = 0; x < clustersList[i].size(); x++) {
      localCluster[clustersList[i][x]] = *runningCluster + THREAD_BLOCKS + 1;
    }
    (*runningCluster)++;
  }

  /**
   **************************************************************************
   * After finilazing the cluster, check the remaining points and
   * insert one point to each of the seedlist
   **************************************************************************
   */

  for (int i = 0; i < THREAD_BLOCKS; i++) {
    while (!unprocessedPoints.empty()) {
      int lastPoint = unprocessedPoints.back();
      unprocessedPoints.pop_back();
      if (localCluster[lastPoint] == UNPROCESSED) {
        localSeedLength[i] = 1;
        localSeedList[i * MAX_SEEDS] = lastPoint;
        break;
      }
    }
  }

  /**
  **************************************************************************
  * FInally, transfer back the CPU memory to GPU and run DBSCAN process
  **************************************************************************
  */

  gpuErrchk(hipMemcpy(d_cluster, localCluster, sizeof(int) * DATASET_COUNT,
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_seedLength, localSeedLength,
                       sizeof(int) * THREAD_BLOCKS, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_seedList, localSeedList,
                       sizeof(int) * THREAD_BLOCKS * MAX_SEEDS,
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMemset(d_collisionMatrix, -1,
                       sizeof(int) * THREAD_BLOCKS * THREAD_BLOCKS));
  gpuErrchk(hipMemset(d_extraCollision, -1,
                       sizeof(int) * THREAD_BLOCKS * EXTRA_COLLISION_SIZE));
  gpuErrchk(hipMemset(d_extraCollisionLength, 0, sizeof(int) * THREAD_BLOCKS));

  if (unprocessedPoints.empty()) {
    int localClusterCount = 0;
    int localNoiseCount = 0;
    for (int i = THREAD_BLOCKS + 1; i < (*runningCluster) + THREAD_BLOCKS + 1;
         i++) {
      bool found = false;
      for (int j = 0; j < DATASET_COUNT; j++) {
        if (localCluster[j] == i) {
          found = true;
          break;
        }
      }
      if (found) {
        localClusterCount++;
      }
    }
    for (int j = 0; j < DATASET_COUNT; j++) {
      if (localCluster[j] == NOISE) {
        localNoiseCount++;
      }
    }
    *clusterCount = localClusterCount;
    *noiseCount = localNoiseCount;

  }

  free(localCluster);
  free(localSeedList);
  free(localRefillSeedList);
  free(localSeedLength);
  free(localRefillSeedLength);
  free(localCollisionMatrix);
  free(localExtraCollision);
  free(localExtraCollisionLength);

  // IF all points has been processed exit DBSCAN
  if (unprocessedPoints.empty()) return true;

  return false;
}

__global__ void DBSCAN(double *dataset, int *cluster, int *seedList,
                       int *seedLength, int *refillSeedList,
                       int *refillSeedLength, int *collisionMatrix,
                       int *extraCollision, int *extraCollisionLength) {
  /**
   **************************************************************************
   * Define shared variables
   **************************************************************************
   */

  __shared__ int pointID;
  __shared__ int neighborBuffer[MINPTS];
  __shared__ int neighborCount;
  __shared__ int chainID;
  __shared__ int point[DIMENSION];
  __shared__ int currentSeedLength;

  /**
   **************************************************************************
   * Get current chain length, and If its zero, exit
   **************************************************************************
   */

  if (threadIdx.x == 0) {
    chainID = blockIdx.x;
    currentSeedLength = seedLength[chainID];
    pointID = seedList[chainID * MAX_SEEDS + currentSeedLength - 1];
  }
  __syncthreads();
  if (currentSeedLength == 0) return;

  __shared__ int processed;
  if (threadIdx.x == 0) {
    processed = false;
    seedLength[chainID] = currentSeedLength - 1;
    if (cluster[pointID] != UNPROCESSED) processed = true;
  }

  __syncthreads();

  if (processed) return;

  if (threadIdx.x == 0) {
    neighborCount = 0;
    for (int x = 0; x < DIMENSION; x++) {
      point[x] = dataset[pointID * DIMENSION + x];
    }
  }
  __syncthreads();

  /**
   **************************************************************************
   * Find the neighbors of the pointID
   * Mark point as candidate if points are more than min points
   * Keep record of left over neighbors in neighborBuffer
   **************************************************************************
   */
  for (int i = threadIdx.x; i < DATASET_COUNT; i = i + THREAD_COUNT) {
    register double comparingPoint[DIMENSION];
    for (int x = 0; x < DIMENSION; x++) {
      comparingPoint[x] = dataset[i * DIMENSION + x];
    }

    register double distance = 0;
    for (int x = 0; x < DIMENSION; x++) {
      distance +=
          (point[x] - comparingPoint[x]) * (point[x] - comparingPoint[x]);
    }

    if (distance <= eps * eps) {
      register int currentNeighborCount = atomicAdd(&neighborCount, 1);
      if (currentNeighborCount >= minPts) {
        MarkAsCandidate(i, chainID, cluster, seedList, seedLength,
                        refillSeedList, refillSeedLength, collisionMatrix,
                        extraCollision, extraCollisionLength);
      } else {
        neighborBuffer[currentNeighborCount] = i;
      }
    }
  }
  __syncthreads();

  /**
   **************************************************************************
   * Mark the left over neighbors in neighborBuffer as cluster member
   * If neighbors are less than minPts, assign pointID with noise
   **************************************************************************
   */

  if (neighborCount >= minPts) {
    cluster[pointID] = chainID;
    for (int i = threadIdx.x; i < minPts; i = i + THREAD_COUNT) {
      MarkAsCandidate(neighborBuffer[i], chainID, cluster, seedList, seedLength,
                      refillSeedList, refillSeedLength, collisionMatrix,
                      extraCollision, extraCollisionLength);
    }
  } else {
    cluster[pointID] = NOISE;
  }

  __syncthreads();

  /**
   **************************************************************************
   * Check Thread length, If it exceeds MAX limit the length
   * As seedlist wont have data beyond its max length
   **************************************************************************
   */

  if (threadIdx.x == 0 && seedLength[chainID] >= MAX_SEEDS) {
    seedLength[chainID] = MAX_SEEDS - 1;
  }

  __syncthreads();

  if (threadIdx.x == 0 && refillSeedLength[chainID] >= REFILL_MAX_SEEDS) {
    refillSeedLength[chainID] = REFILL_MAX_SEEDS - 1;
  }
  __syncthreads();
}

__device__ void MarkAsCandidate(int neighborID, int chainID, int *cluster,
                                int *seedList, int *seedLength,
                                int *refillSeedList, int *refillSeedLength,
                                int *collisionMatrix, int *extraCollision,
                                int *extraCollisionLength) {
  /**
  **************************************************************************
  * Get the old cluster state of the neighbor
  * If the state is unprocessed, assign it with chainID
  **************************************************************************
  */
  register int oldState = atomicCAS(&cluster[neighborID], UNPROCESSED, chainID);

  /**
   **************************************************************************
   * For unprocessed old state of neighbors, add them to seedlist and
   * refill seedlist
   **************************************************************************
   */
  if (oldState == UNPROCESSED) {
    register int sl = atomicAdd(&(seedLength[chainID]), 1);
    if (sl < MAX_SEEDS) {
      seedList[chainID * MAX_SEEDS + sl] = neighborID;
    } else {
      register int rsl = atomicAdd(&(refillSeedLength[chainID]), 1);
      if (rsl < REFILL_MAX_SEEDS) {
        refillSeedList[chainID * REFILL_MAX_SEEDS + rsl] = neighborID;
      }
    }
  }

  /**
   **************************************************************************
   * If the old state of neighbor is not noise, not member of chain and cluster
   * is within THREADBLOCK, maek the collision between old and new state
   **************************************************************************
   */
  else if (oldState != NOISE && oldState != chainID &&
           oldState < THREAD_BLOCKS &&
           collisionMatrix[oldState * THREAD_BLOCKS + chainID] == -1 &&
           collisionMatrix[oldState * THREAD_BLOCKS + chainID] == -1) {
    if (chainID > oldState) {
      collisionMatrix[oldState * THREAD_BLOCKS + chainID] = 1;
    } else {
      collisionMatrix[chainID * THREAD_BLOCKS + oldState] = 1;
    }
  }

  /**
   **************************************************************************
   * If the old state is noise, assign it to chainID cluster
   **************************************************************************
   */
  else if (oldState == NOISE) {
    oldState = atomicCAS(&(cluster[neighborID]), NOISE, chainID);
  } else if (oldState >= THREAD_BLOCKS) {
    register int cl = atomicAdd(&(extraCollisionLength[chainID]), 1);
    extraCollision[chainID * EXTRA_COLLISION_SIZE + cl] = oldState;

    bool alreadyThere = false;
    for (int k = 0; k < cl; k++) {
      if (oldState == extraCollision[chainID * EXTRA_COLLISION_SIZE + k]) {
        alreadyThere = true;
        break;
      }
    }
    if (alreadyThere) {
      cl = atomicSub(&(extraCollisionLength[chainID]), 1);
    }
  }
}

/**
 **************************************************************************
 * Import dataset from file.
 **************************************************************************
 */
int ImportDataset(char const *fname, double *dataset) {
  FILE *fp = fopen(fname, "r");
  if (!fp) {
    printf("Unable to open file\n");
    return (1);
  }

  char buf[4096];
  unsigned long int cnt = 0;
  while (fgets(buf, 4096, fp) && cnt < DATASET_COUNT * DIMENSION) {
    char *field = strtok(buf, ",");
    long double tmp;
    sscanf(field, "%Lf", &tmp);
    dataset[cnt] = tmp;
    cnt++;

    while (field) {
      field = strtok(NULL, ",");

      if (field != NULL) {
        long double tmp;
        sscanf(field, "%Lf", &tmp);
        dataset[cnt] = tmp;
        cnt++;
      }
    }
  }
  fclose(fp);
  return 0;
}