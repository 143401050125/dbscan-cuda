
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <vector>
#include <algorithm> 
#include <string.h>
#include <fstream>
#include <iostream>
#include <string>
#include <map>

#define DATASET_COUNT 100
#define DIMENSION 2

#define MAX_SEEDS 32

#define THREAD_BLOCKS 8
#define THREAD_COUNT 16

#define UNPROCESSED 0
#define NOISE -1

#define MINPTS 5
#define EPS 0.4

using namespace std;

int importDataset(char *fname, double *dataset);

bool monitorSeedPoints(int * d_clusters, int * d_seedList, int * d_currentSeedLength, bool * collisionMatrix, vector<int> &pointsRemaining, int * clusterCount);

__global__ void DBSCAN(double * dataset, int * clusters, int * seedList, int * currentSeedLength, bool * collisionMatrix);

__device__ void processObject(int pointID, int comparingPointID, int * neighborsCount, int* neighbors, int * seedList, double * dataset);

__device__ void markAsCandidate(int neighborID, int chainID, int * clusters, bool * collisionMatrix, int * currentSeedLength, int * seedList) ;

int main(int argc, char *argv[]) {
  if (argc!=2){
	printf("Provide dataset file in the argument");
	return 0;
	}
	char inputFname[500];
	strcpy(inputFname,argv[1]);
   
  printf("Dataset file: %s\n",inputFname);
  
  double * h_dataset = (double*)malloc(sizeof(double)*DATASET_COUNT*DIMENSION);
  int ret = importDataset(inputFname, h_dataset);
  if (ret==1){
    printf("\nError importing the dataset");
    return 0;
  }

  // CPU Variable declarations
  vector<int> pointsRemaining;

  // GPU Variable declarations
  double * d_dataset;
  int * d_clusters;
  int * d_seedList;
  int * d_currentSeedLength;
  bool * d_collisionMatrix;

  // GPU Memory allocation
  hipMalloc((void **) &d_dataset, sizeof(double)*DATASET_COUNT*DIMENSION);
  hipMalloc((void **) &d_clusters, sizeof(int)*DATASET_COUNT);
  hipMalloc((void **) &d_seedList, sizeof(int)*THREAD_BLOCKS*MAX_SEEDS);
  hipMalloc((void **) &d_currentSeedLength, sizeof(int)*THREAD_BLOCKS);
  hipMalloc((void **) &d_collisionMatrix,  sizeof(bool)*THREAD_BLOCKS*THREAD_BLOCKS);

  // Initializing data in dataset
  for (int x = 0; x < DATASET_COUNT; x++){
    pointsRemaining.push_back(x);
  }
  
  printf("Dataset has %d data. \n", pointsRemaining.size());

  // Initialize GPU variables with data
  hipMemset(d_seedList, -1, sizeof(int) * THREAD_BLOCKS * MAX_SEEDS);
  hipMemset(d_currentSeedLength, 0, sizeof(int) * THREAD_BLOCKS);
  hipMemset(d_clusters, UNPROCESSED, sizeof(int) * DATASET_COUNT);
  hipMemset(d_collisionMatrix, false, sizeof(bool) * THREAD_BLOCKS * THREAD_BLOCKS);
  hipMemcpy(d_dataset, h_dataset, sizeof(double) * DATASET_COUNT * DIMENSION, hipMemcpyHostToDevice);

  // Start the DBSCAN process between CPU AND GPU
  int clusterCount = 0;
  bool exit = false;
  int count = 0;

  while(!exit) {

    bool completed = monitorSeedPoints(d_clusters, d_seedList, d_currentSeedLength, d_collisionMatrix, pointsRemaining, &clusterCount);

    if(completed) {
      exit = true;
    }

    /******************************************************
     *  Run DBSCAN Kernel
     ******************************************************
    */

    dim3 GRID(THREAD_BLOCKS, 1);
    dim3 BLOCK(THREAD_COUNT,1);
    
    DBSCAN<<<GRID, BLOCK>>>(d_dataset, d_clusters, d_seedList, d_currentSeedLength, d_collisionMatrix);

    /******************************************************
     *  /end Run DBSCAN Kernel
     ******************************************************
    */
    printf("Number of clusters %d, points remaining: %d\n", clusterCount, pointsRemaining.size());
    if(exit) break;
  }
}

bool monitorSeedPoints(int * d_clusters, int * d_seedList, int * d_currentSeedLength, bool * d_collisionMatrix, vector<int> &pointsRemaining, int * clusterCount) {
  
  int * localSeedCount = (int *)malloc(sizeof(int) * THREAD_BLOCKS);

  memset(localSeedCount, 0, sizeof(int)*THREAD_BLOCKS);
  hipMemcpy(localSeedCount, d_currentSeedLength, sizeof(int) *THREAD_BLOCKS, hipMemcpyDeviceToHost);

  bool completeSeedListFirst = false;
  for(int i = 0; i < THREAD_BLOCKS; i++) {
    if(localSeedCount[i] > 0) {
      completeSeedListFirst = true;
      break;
    }
  }

  if(completeSeedListFirst) {
    return false;
  }

  int * localClusters = (int *)malloc(sizeof(int) * DATASET_COUNT);
  hipMemcpy(localClusters, d_clusters, sizeof(int) * DATASET_COUNT, hipMemcpyDeviceToHost);

  bool* localCollisionMatrix = (bool *)malloc(sizeof(bool) * THREAD_BLOCKS*THREAD_BLOCKS);

  hipMemcpy(localCollisionMatrix, d_collisionMatrix, sizeof(bool) * THREAD_BLOCKS * THREAD_BLOCKS, hipMemcpyDeviceToHost);

  map<int, int> clusterMap;
  for(int x = 0; x < THREAD_BLOCKS; x++) {
    for(int y = 0; y < THREAD_BLOCKS; y++) {
      if(localCollisionMatrix[THREAD_BLOCKS*x + y] || localCollisionMatrix[THREAD_BLOCKS*y +x]) {
        clusterMap[y] = clusterMap[x];
      }
    }
  }

  vector<vector<int>> clustersList(THREAD_BLOCKS, vector<int>());
  for(int i = 0; i < DATASET_COUNT; i++) {
    if(localClusters[i] > 0 && localClusters[i] < THREAD_BLOCKS) {
      clustersList[clusterMap[localClusters[i]]].push_back(i);
    }
  }
  
  for(int i = 0; i < clustersList.size(); i++) {
    if(clustersList[i].size() != 0) (*clusterCount)++;
    for(int j = 0; j < clustersList[i].size(); j++) {
      localClusters[clustersList[i][j]] = *clusterCount + THREAD_BLOCKS;
    }
  }

  int * localSeedList = (int *)malloc(sizeof(int) * THREAD_BLOCKS*MAX_SEEDS);

  hipMemcpy(localSeedList, d_seedList, sizeof(int) * THREAD_BLOCKS * MAX_SEEDS, hipMemcpyDeviceToHost);

  // Initialize with first seed point
  int processedCount = 0;
  for(int i = 0; i < THREAD_BLOCKS; i++) {
    bool processed = true;
    while(!pointsRemaining.empty()) {
      int point = pointsRemaining.back();
      pointsRemaining.pop_back();
      if(localClusters[point] == UNPROCESSED) {
        localSeedList[i*MAX_SEEDS] = point;
        localSeedCount[i] = 1;
        processed = false;
        break;
      }
    }
    if(processed) {
      processedCount++;
    }
  }

  if(processedCount == THREAD_BLOCKS) {
    return true;
  }

  hipMemset(d_collisionMatrix, false, sizeof(bool) * THREAD_BLOCKS * THREAD_BLOCKS);

  hipMemcpy(d_clusters, localClusters, sizeof(int)* DATASET_COUNT, hipMemcpyHostToDevice);

  hipMemcpy(d_seedList, localSeedList, sizeof(int)* THREAD_BLOCKS* MAX_SEEDS, hipMemcpyHostToDevice);

  hipMemcpy(d_currentSeedLength, localSeedCount, sizeof(int)* THREAD_BLOCKS, hipMemcpyHostToDevice);

  return false;
}



__global__ void DBSCAN(double * dataset, int * clusters, int * seedList, int * currentSeedLength, bool * collisionMatrix){

  __shared__ int pointID;

  __shared__ int neighbors[MAX_SEEDS];

  __shared__ int neighborsCount;

  int chainID = blockIdx.x;

  int seedLength = currentSeedLength[chainID];

  // If seedlength is 0 then exit
  if(seedLength == 0) return;

  pointID = seedList[chainID*MAX_SEEDS + seedLength - 1];  

  // If the point is already processed then exit
  __shared__ bool processed;
  processed = false;

  if(threadIdx.x == 0) {
    currentSeedLength[chainID] = currentSeedLength[chainID] - 1;
    if(clusters[pointID] != UNPROCESSED) processed = true;
  }
  __syncthreads();
  if(processed) return;

  for(int i = threadIdx.x; i < DATASET_COUNT; i=i+THREAD_COUNT) {
    processObject(pointID, i, &neighborsCount, neighbors, seedList, dataset);
  }

  __syncthreads();

  if(neighborsCount >= MINPTS) {
    clusters[pointID] = chainID;
    for(int i = threadIdx.x; i < neighborsCount; i=i+THREAD_COUNT) {
      markAsCandidate(neighbors[i], chainID, clusters, collisionMatrix, currentSeedLength, seedList);
    }
  } else {
    clusters[pointID] = NOISE;
  }

  __syncthreads();

  if(threadIdx.x == 0 && currentSeedLength[chainID] >= MAX_SEEDS) {
    currentSeedLength[chainID] = MAX_SEEDS - 1;
  }
}

__device__ void processObject(int pointID, int comparingPointID, int * neighborsCount, int* neighbors, int * seedList, double * dataset) {
  register double comparingPoint[DIMENSION], point[DIMENSION];
  for(int i = 0; i < DIMENSION; i++) {
    point[i] = seedList[pointID*MAX_SEEDS + i];
    comparingPoint[i] = dataset[comparingPointID*DIMENSION + i];
  }

  register double distance = 0;
  for(int x = 0; x < DIMENSION; x++) {
    distance += (comparingPoint[x] - point[x])*(comparingPoint[x] - point[x]);
  }
  distance = sqrtf(distance);

  if(distance <= EPS) {
    register int currentNeighbor = atomicAdd(neighborsCount, 1);
    neighbors[currentNeighbor] = comparingPointID;
  }
}

__device__ void markAsCandidate(int neighborID, int chainID, int * clusters, bool * collisionMatrix, int * currentSeedLength, int * seedList) {

  register int oldState = atomicCAS(&(clusters[neighborID]), UNPROCESSED, chainID);

  if(oldState == UNPROCESSED) {
    register int seedLength = atomicAdd(&(currentSeedLength[chainID]), 1); 
    if(seedLength < MAX_SEEDS) {
      seedList[chainID*MAX_SEEDS + seedLength] = neighborID;
    }
  } else if (oldState == NOISE) {
    
    atomicCAS(&(clusters[neighborID]), NOISE, chainID);

  } else if(oldState != NOISE && oldState != chainID && oldState < THREAD_BLOCKS) {

    if(oldState < chainID) {
      collisionMatrix[oldState*THREAD_BLOCKS + chainID] = true;
    } else {
      collisionMatrix[chainID*THREAD_BLOCKS + oldState] = true;
    }
  }
}


int importDataset(char * fname, double * dataset){
  FILE *fp = fopen(fname, "r");
  if (!fp) {
      printf("Unable to open file\n");
      return(1);
  }
  char buf[4096];
  unsigned long int cnt=0;
  while (fgets(buf, 4096, fp) && cnt < DATASET_COUNT) {
      char *field = strtok(buf, ",");
      double tmp;
      sscanf(field,"%lf",&tmp);
      dataset[cnt]=tmp;
      cnt++;
      while (field) {
        field = strtok(NULL, ",");
        if (field!=NULL) {
          double tmp;
          sscanf(field,"%lf",&tmp);
          dataset[cnt] = tmp;
          cnt++;
        }   
      }
  }
  fclose(fp);
  return 0;
}